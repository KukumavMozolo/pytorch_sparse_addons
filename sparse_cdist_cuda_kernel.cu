#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "math.h"


template <typename scalar_t> __device__ void cpy_array(scalar_t* from, scalar_t* to, int start, int end)
{
  int counter = 0;
  for (int i=start; i<end; i++){
    to[counter]=from[i];
    counter++;
  }
}




template <typename scalar_t>
__global__ void sparse_cdist_cuda_kernel(
    const int64_t* __restrict__ a_rowptr,
    const int64_t* __restrict__ a_col,
    scalar_t* __restrict__ a_value,
    int64_t* __restrict__ b_rowptr,
    int64_t* __restrict__ b_col,
    scalar_t* __restrict__ b_value,
    scalar_t* __restrict__ output,
    int dim_a,
    int dim_b) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < dim_a && j < dim_b){
    const int start_i = a_rowptr[i];
    const int end_i = a_rowptr[i+1];
    const int start_j = b_rowptr[j];
    const int end_j = b_rowptr[j+1];

    scalar_t distance = 0.0;

    scalar_t *b_value_remainder = new scalar_t[end_j-start_j];
    cpy_array<scalar_t>(b_value, b_value_remainder, start_j, end_j);

    for (int ii = start_i; ii < end_i; ii ++){
      int col_index_i = a_col[ii];
      auto value_i = a_value[ii];
      bool not_matched_i = true;
      int counter = 0;
      for (int jj = start_j; jj < end_j; jj ++){
        int col_index_j = b_col[jj];
        auto value_j = b_value[jj];

        if (col_index_i == col_index_j){
          auto t = (value_i - value_j);
          t *=t;
          distance += t;
          not_matched_i = false;
          b_value_remainder[counter] = 0.0;
        }
        counter++;
      }
      if(not_matched_i){
        distance +=(value_i*value_i);
      }
    }
    for (int jj = 0; jj < end_j- start_j; jj ++){
      distance +=(b_value_remainder[jj]*b_value_remainder[jj]);
    }
    distance = sqrt(distance);
    output[i*dim_b + j] = distance;

  }
}


template <typename scalar_t>
__global__ void sparse_cdist_bw_cuda_kernel(
    const int64_t* __restrict__ a_row,
    const int64_t* __restrict__ a_col,
    scalar_t* __restrict__ a_value,
    int64_t* __restrict__ b_row,
    int64_t* __restrict__ b_col,
    scalar_t* __restrict__ b_value,
    scalar_t* __restrict__ grad_out,
    scalar_t* __restrict__ distances,
    scalar_t* __restrict__ grad,
    int b_col_size,
    int dim_distance_a
  ) {
  const int m = blockIdx.x * blockDim.x + threadIdx.x;
  const int n = blockIdx.y * blockDim.y + threadIdx.y;

  auto a_mn_value_grad = a_value[m];
  
  for(int j=0; j < b_col_size, j++){
    if(b_col == n){
      scalar_t b_val_jn= b_value[j];
      scalar_t aggregator = 0;
      for(int i=0; i < dim_distance_a, i++){
        aggregator += grad_out[i*dim_distance_a + j] * distances[i*dim_distance_a + j];
      }
      a_mn_value_grad -= b_val_jn * aggregator;
    }
  }
  grad[m] = a_mn_value_grad;
}




at::Tensor sparse_cdist_cuda(
    torch::Tensor a_rowptr_data,
    torch::Tensor a_col_data,
    torch::Tensor a_value_data,
    torch::Tensor b_rowptr_data,
    torch::Tensor b_col_data,
    torch::Tensor b_value_data,
    int dim_a,
    int dim_b
    ) {

  std::vector<int64_t> vec;
  vec.push_back(dim_a);
  vec.push_back(dim_b);
  auto options = a_value_data.options();
  torch::Tensor output = torch::zeros(vec,options = options);

  
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(a_rowptr_data.size(0)+1 / threadsPerBlock.x, b_rowptr_data.size(0)+1 / threadsPerBlock.y);
  AT_DISPATCH_FLOATING_TYPES(a_value_data.scalar_type(), "sparse_cdist_cuda", ([&] {
    sparse_cdist_cuda_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
        a_rowptr_data.data_ptr<int64_t>(),
        a_col_data.data_ptr<int64_t>(),
        a_value_data.data_ptr<scalar_t>(),
        b_rowptr_data.data_ptr<int64_t>(),
        b_col_data.data_ptr<int64_t>(),
        b_value_data.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        a_rowptr_data.size(0),
        b_rowptr_data.size(0));

  }));

  return output;
}

torch::Tensor sparse_cdist_bw_cuda(
    torch::Tensor a_rowptr_data,
    torch::Tensor a_col_data,
    torch::Tensor a_value_data,
    torch::Tensor b_rowptr_data,
    torch::Tensor b_col_data,
    torch::Tensor b_value_data,
    torch::Tensor grad_out,
    torch::Tensor distance,
    int dim_a,
    int dim_b
    ) {

  torch::Tensor grad_a = torch::zeros_like(grad_out, grad_out.options());
  std::cout << "grad_a after creation is: " << grad_a;

  
  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(a_rowptr_data.size(0)+1 / threadsPerBlock.x, b_rowptr_data.size(0)+1 / threadsPerBlock.y);
  AT_DISPATCH_FLOATING_TYPES(a_value_data.scalar_type(), "sparse_cdist_bw_cuda", ([&] {
    sparse_cdist_bw_cuda_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
        a_rowptr_data.data_ptr<int64_t>(),
        a_col_data.data_ptr<int64_t>(),
        a_value_data.data_ptr<scalar_t>(),
        b_rowptr_data.data_ptr<int64_t>(),
        b_col_data.data_ptr<int64_t>(),
        b_value_data.data_ptr<scalar_t>(),
        grad_out.data_ptr<scalar_t>(),
        distance.data_ptr<scalar_t>(),
        grad_a.data_ptr<scalar_t>(),
        a_rowptr_data.size(0),
        b_rowptr_data.size(0));

  }));

  return grad_a;
}